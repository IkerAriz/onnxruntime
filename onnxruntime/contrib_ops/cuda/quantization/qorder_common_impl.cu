#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation. All rights reserved.
// Licensed under the MIT License.
#include "contrib_ops/cuda/quantization/qorder_unary_ops_impl.h"
#include "core/providers/cuda/cu_inc/common.cuh"
#include "core/providers/cuda/shared_inc/cuda_utils.h"

namespace onnxruntime {
namespace contrib {
namespace cuda {

using namespace onnxruntime::cuda;

/*
* Utilility types and functions
*/
struct __half4 {
  __half2 xy;
  __half2 zw;
};

union U4S2 {
  unsigned u4;
  short2 s2;
};

template <typename CharVecT>
struct HalfVecMap {
}; 

template struct HalfVecMap<char2> {
typedef __half2 MappedType;
};

template struct HalfVecMap<char4> {
typedef __half4 MappedType;
};

template <typename CharVectT>
struct QuantizeHalfVec {
    HlafVecMap<CharVecT>::MappedType operator(CharVecT chars, __half2 inverse_scale2);
};

template 
struct QuantizeHalfVec<char2> {
    HlafVecMap<CharVecT> operator(CharVecT chars, __half2 inverse_scale2);
};


__device__ inline int8_t quantize_float_s8(const float& val, const float& inverse_scale) {
  float dqval = fmaxf(fminf(127.0f, val * inverse_scale), -128.0f);
  return static_cast<int8_t>(__float2int_rn(dqval));
}

__device__ inline char4 quantize_half4_char4(__half4 val4, const __half2 inverse_scale2) {
  val4.xy *= inverse_scale2;
  val4.zw *= inverse_scale2;
  U4S2 shortxy, shortzw;
  shortxy.s2.x = __half2short_rn(__low2half(val4.xy));
  shortzw.s2.x = __half2short_rn(__low2half(val4.zw));
  shortxy.s2.y = __half2short_rn(__high2half(val4.xy));
  shortzw.s2.y = __half2short_rn(__high2half(val4.zw));
  shortxy.u4 = __vmaxs2(__vmins2(shortxy.u4, 0x007F007F), 0xFF80FF80);
  shortzw.u4 = __vmaxs2(__vmins2(shortzw.u4, 0x007F007F), 0xFF80FF80);
  return char4{(char)shortxy.s2.x, (char)shortxy.s2.y, (char)shortzw.s2.x, (char)shortzw.s2.y};
}

__device__ inline __half4 deqantize_char4_half4(const char4 ch4, const __half2 scale2) {
  return {scale2 * __half2(__short2half_rn(ch4.x), __short2half_rn(ch4.y)),
          scale2 * __half2(__short2half_rn(ch4.z), __short2half_rn(ch4.w))};
}

template <typename T>
__inline__ __device__ T
WarpReduceSum(T val) {
  val += __shfl_xor_sync(0xFFFFFFFF, val, 1);
  val += __shfl_xor_sync(0xFFFFFFFF, val, 2);
  val += __shfl_xor_sync(0xFFFFFFFF, val, 4);
  val += __shfl_xor_sync(0xFFFFFFFF, val, 8);
  val += __shfl_xor_sync(0xFFFFFFFF, val, 16);
  return val;
}


/************************************************************************
 * Quantize Routines:
 *   - OrderRow (fp16/32) to OrderCol32 (cols % 32 == 0)
 ************************************************************************/

// source matrix block 32 x 32, each thread handle 4 int8 items, so:
// thread block size should be (8 cols_in_4, 32 rows, 1)
// grid size ((cols + 31) / 32, (rows + 31) / 32), batch)
__global__ void
QOrderQuantizeHalfRowToCol32Kernel(const __half* __restrict__ src, size_t src_batch_stride,
                                   int8_t* __restrict__ dst, size_t dst_batch_stride,
                                   const __half2 inverse_scale2, unsigned rows, unsigned cols) {
  unsigned int c = (blockIdx.x * blockDim.x + threadIdx.x) << 2;
  unsigned int r = blockIdx.y * blockDim.y + threadIdx.y;
  if (c < cols && r < rows) {
    const size_t src_index = (src_batch_stride * blockIdx.z) + (r * cols + c);
    const size_t dst_index = (dst_batch_stride * blockIdx.z) + ((c & 0xffffffe0) * rows + (r << 5) + (c & 0x1F));
    __half4 const src_val4 = *((const __half4*)(src + src_index));
    *(char4*)(dst + dst_index) = quantize_half4_char4(src_val4, inverse_scale2);
  }
}

// cols could be divide by 32
void QOrderQuantizeRowToCol32(hipStream_t stream, const hipDeviceProp_t& /*device_prop*/,
                              const __half* src, int8_t* dst, float scale,
                              unsigned batch, unsigned rows, unsigned cols) {
  if (cols & 0x1f) {
    throw std::runtime_error("cols can not divide by 32!");
  }

  __half2 inverse_scale2 = __float2half2_rn(1.0f / scale);
  dim3 threads(8, 32, 1);
  dim3 blocks(cols / 32, (rows + 31) / 32, batch);
  size_t stride = (size_t)rows * cols;
  QOrderQuantizeHalfRowToCol32Kernel<<<blocks, threads, 0, stream>>>(src, stride, dst, stride, inverse_scale2, rows, cols);
}

// source matrix block (32 x ElementsPerThread) x 32, each thread handle ElementsPerThread elements items, so:
// thread block size should be (32 cols, 32 rows, 1)
// grid size ((cols + 32*ElementsPerThread - 1) / (32 * ElementsPerThread), (rows + 31) / 32), batch)
template <unsigned ElementsPerThread = 4>
__global__ void
QOrderQuantizeFloatRowToCol32Kernel(const float* __restrict__ src, size_t src_batch_stride,
                                    int8_t* __restrict__ dst, size_t dst_batch_stride,
                                    const float inverse_scale, unsigned rows, unsigned cols) {
  unsigned int r = blockIdx.y * blockDim.y + threadIdx.y;
  static constexpr unsigned kColsPerIncrement = 32; // it is the blockDim.x
  if (r < rows) {
    unsigned int c = blockIdx.x * (kColsPerIncrement * ElementsPerThread) + threadIdx.x;
    size_t src_index = (src_batch_stride * blockIdx.z) + (r * cols + c);
    size_t dst_index = (dst_batch_stride * blockIdx.z) + ((c & 0xffffffe0) * rows + (r << 5) + (c & 0x1f));

#pragma unroll
    for (int i = 0; i < ElementsPerThread; i++) {
      if (c < cols) {
        *(dst + dst_index) = quantize_float_s8(*(src + src_index), inverse_scale);
        c += kColsPerIncrement;
        src_index += kColsPerIncrement;
        dst_index += rows * kColsPerIncrement;
      }
    }
  }
}

// cols could be divide by 32
void QOrderQuantizeRowToCol32(hipStream_t stream, const hipDeviceProp_t& /*device_prop*/,
                              const float* src, int8_t* dst, float scale,
                              unsigned batch, unsigned rows, unsigned cols) {
  if (cols & 0x1f) {
    throw std::runtime_error("cols can not divide by 32!");
  }

  constexpr unsigned kElementsPerThread = 4;
  float inverse_scale = 1.0f / scale;
  dim3 threads(32, 32, 1);
  dim3 blocks((cols + (32 * kElementsPerThread - 1)) / (kElementsPerThread * 32), (rows + 31) / 32, batch);
  size_t stride = (size_t)rows * cols;
  QOrderQuantizeFloatRowToCol32Kernel<<<blocks, threads, 0, stream>>>(src, stride, dst, stride, inverse_scale, rows, cols);
}

/************************************************************************
 * Quantize Routines:
 *   - fp16/32 input, do no care order
 ************************************************************************/

// thread block size should be (256 (elements in 4), 1, 1)
// grid size ((N + 1023) / 1024, 1, 1)
template <typename TCharVec> // char, char2, char4, 
__global__ void
QOrderQuantizeHalfKernel(const __half* __restrict__ src, int8_t* __restrict__ dst,
                         const __half2 inverse_scale2, size_t N) {
  typedef HalfVecMap<TCharVec>::MappedType THalfVec;
  size_t index = ((size_t)blockIdx.x * blockDim.x + threadIdx.x) * sizeof(TCharVec);
  if (index < N) {
    TCharVec const src_vals = *((const TCharVec*)(src + index));
    *(THalfVec*)(dst + index) = quantize_half4_char4(src_val4, inverse_scale2);
  }
}

void QOrderQuantize(hipStream_t stream, const hipDeviceProp_t& /* device_prop */,
                    const __half* src, int8_t* dst, float scale, size_t N) {
  if (N & 0x1fLL) {
    throw std::runtime_error("N can not divide by 32!");
  }

  __half2 inverse_scale2 = __float2half2_rn(1.0f / scale);
  dim3 threads(256, 1, 1);
  dim3 blocks((N + 1023) / 1024, 1, 1);
  QOrderQuantizeHalfKernel<<<blocks, threads, 0, stream>>>(src, dst, inverse_scale2, N);
}

// thread block size should be (256 (elements in 4), 1, 1)
// grid size ((N + 1023) / 1024, 1, 1)
template <unsigned ElementsPerThread = 4>
__global__ void
QOrderQuantizeFloatKernel(const float* __restrict__ src, int8_t* __restrict__ dst,
                          const float inverse_scale, size_t N) {
  size_t index = (size_t)blockIdx.x * blockDim.x * ElementsPerThread + threadIdx.x;
#pragma unroll
  for (int i = 0; i < ElementsPerThread; i++) {
    if (index < N) {
      *(dst + index) = quantize_float_s8(*(src + index), inverse_scale);
      index += blockDim.x;
    }
  }
}

void QOrderQuantize(hipStream_t stream, const hipDeviceProp_t& device_prop,
                    const float* src, int8_t* dst, float scale, size_t N) {
  if (N & 0x1f) {
    throw std::runtime_error("N can not divide by 32!");
  }

  static constexpr unsigned kElementsPerThread = 4;
  float inverse_scale = 1.0f / scale;
  dim3 threads(256, 1, 1);
  dim3 blocks((N + (threads.x * kElementsPerThread - 1)) / (threads.x * kElementsPerThread), 1, 1);
  QOrderQuantizeFloatKernel<kElementsPerThread><<<blocks, threads, 0, stream>>>(src, dst, inverse_scale, N);
}

/************************************************************************
 * Dequantize Routines:
 *   - Col32 to OrderRow (fp16/32) (cols % 32 == 0)
 ************************************************************************/

// target matrix block 32 x 32, each thread handle 4 int8 items, so:
// thread block size should be (8 cols_in_4, 32 rows, 1)
// grid size ((cols + 31) / 32, (rows + 31) / 32), batch)
__global__ void
QOrderDequantizeCol32ToHalfRowKernel(const int8_t* __restrict__ src, size_t src_batch_stride,
                                     __half* __restrict__ dst, size_t dst_batch_stride,
                                     const __half2 scale2, unsigned rows, unsigned cols) {
  unsigned int c = (blockIdx.x * blockDim.x + threadIdx.x) << 2;
  unsigned int r = blockIdx.y * blockDim.y + threadIdx.y;
  if (c < cols && r < rows) {
    const size_t dst_index = (dst_batch_stride * blockIdx.z) + (r * cols + c);
    const size_t src_index = (src_batch_stride * blockIdx.z) + ((c & 0xffffffe0) * rows + (r << 5) + (c & 0x1F));
    const char4 src_ch4 = *((const char4*)(src + src_index));
    *(__half4*)(dst + dst_index) = deqantize_char4_half4(src_ch4, scale2);
  }
}

// cols could be divide by 32
void QOrderDequantizeCol32ToRow(hipStream_t stream, const hipDeviceProp_t& /*device_prop*/,
                                const int8_t* src, __half* dst, float scale,
                                unsigned batch, unsigned rows, unsigned cols) {
  if (cols & 0x1f) {
    throw std::runtime_error("cols can not divede by 32");
  }

  __half2 scale2 = __float2half2_rn(scale);
  dim3 threads(8, 32, 1);
  dim3 blocks(cols / 32, (rows + 31) / 32, batch);
  size_t stride = (size_t)rows * cols;
  QOrderDequantizeCol32ToHalfRowKernel<<<blocks, threads, 0, stream>>>(src, stride, dst, stride, scale2, rows, cols);
}

// target matrix block 32 x 32, each thread handle 1 items, so:
// thread block size should be (32, 32 rows, 1)
// grid size ((cols / 32), (rows + 31) / 32), batch)
__global__ void
QOrderDequantizeCol32ToFloatRowKernel(const int8_t* __restrict__ src, size_t src_batch_stride,
                                     float* __restrict__ dst, size_t dst_batch_stride,
                                     float scale, unsigned rows, unsigned cols) {
  unsigned int c = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int r = blockIdx.y * blockDim.y + threadIdx.y;
  if (c < cols && r < rows) {
    const size_t dst_index = (dst_batch_stride * blockIdx.z) + (r * cols + c);
    const size_t src_index = (src_batch_stride * blockIdx.z) + ((c & 0xffffffe0) * rows + (r << 5) + (c & 0x1F));
    dst[dst_index] = scale * static_cast<float>(src[src_index]);
  }
}

void QOrderDequantizeCol32ToRow(hipStream_t stream, const hipDeviceProp_t& /*device_prop*/,
                                const int8_t* src, float* dst, float scale,
                                unsigned batch, unsigned rows, unsigned cols) {
  if (cols & 0x1f) {
    throw std::runtime_error("cols can not divede by 32");
  }
  dim3 threads(32, 32, 1);
  dim3 blocks(cols / 32, (rows + 31) / 32, batch);
  size_t stride = (size_t)rows * cols;
  QOrderDequantizeCol32ToFloatRowKernel<<<blocks, threads, 0, stream>>>(src, stride, dst, stride, scale, rows, cols);
}

/************************************************************************
 * Dequantize Routines:
 *   - fp16/32 output, do no care order
 ************************************************************************/

// thread block size should be (256 (elements in 4), 1, 1)
// grid size ((N + 1023) / 1024, 1, 1)
template <unsigned ElementsPerThread = 4>
__global__ void
QOrderDeuantizeHalfKernel(const int8_t* __restrict__ src, __half* __restrict__ dst,
                         const __half2 scale2, size_t N) {
  size_t index = ((size_t)blockIdx.x * blockDim.x + threadIdx.x) << 2;

#pragma unroll
  for (int i = 0; i < ElementsPerThread; i++) {
    if (index < N) {
      __half4 const src_val4 = *((const __half4*)(src + index));
      *(char4*)(dst + index) = quantize_half4_char4(src_val4, inverse_scale2);
      index += blockDim.x;
    }
  }
}

void QOrderQuantize(hipStream_t stream, const hipDeviceProp_t& /* device_prop */,
                    const __half* src, int8_t* dst, float scale, size_t N) {
  if (N & 0x1fLL) {
    throw std::runtime_error("N can not divide by 32!");
  }

  __half2 inverse_scale2 = __float2half2_rn(1.0f / scale);
  dim3 threads(256, 1, 1);
  dim3 blocks((N + 1023) / 1024, 1, 1);
  QOrderQuantizeHalfKernel<<<blocks, threads, 0, stream>>>(src, dst, inverse_scale2, N);
}

// thread block size should be (256 (elements in 4), 1, 1)
// grid size ((N + 1023) / 1024, 1, 1)
template <unsigned ElementsPerThread = 4>
__global__ void
QOrderQuantizeFloatKernel(const float* __restrict__ src, int8_t* __restrict__ dst,
                          const float inverse_scale, size_t N) {
  size_t index = (size_t)blockIdx.x * blockDim.x * ElementsPerThread + threadIdx.x;
#pragma unroll
  for (int i = 0; i < ElementsPerThread; i++) {
    if (index < N) {
      *(dst + index) = quantize_float_s8(*(src + index), inverse_scale);
      index += blockDim.x;
    }
  }
}

void QOrderQuantize(hipStream_t stream, const hipDeviceProp_t& device_prop,
                    const float* src, int8_t* dst, float scale, size_t N) {
  if (N & 0x1f) {
    throw std::runtime_error("N can not divide by 32!");
  }

  static constexpr unsigned kElementsPerThread = 4;
  float inverse_scale = 1.0f / scale;
  dim3 threads(256, 1, 1);
  dim3 blocks((N + (threads.x * kElementsPerThread - 1)) / (threads.x * kElementsPerThread), 1, 1);
  QOrderQuantizeFloatKernel<kElementsPerThread><<<blocks, threads, 0, stream>>>(src, dst, inverse_scale, N);
}

static constexpr unsigned QORDER_LAYERNORM_ROWS_PER_BLOCK = 8;  // 4, 8, 16, ...
// block_size = (32, QORDER_LAYERNORM_ROWS_PER_BLOCK, 1)
// grid_size = ((rows + QORDER_LAYERNORM_ROWS_PER_BLOCK - 1) / QORDER_LAYERNORM_ROWS_PER_BLOCK, batch, 1)
__global__ void
QOrderLayerNormKernel(const int8_t* __restrict__ src, const float src_scale, int8_t* __restrict__ dst, const float dst_scale,
                      const __half* __restrict__ gamma, const __half* __restrict__ beta, const float epsilon,
                      const unsigned rows, const unsigned cols) {
  int32_t sum = 0;
  int32_t square_sum = 0;
  unsigned r = blockIdx.x * QORDER_LAYERNORM_ROWS_PER_BLOCK + threadIdx.y;
  if (rows <= r) return;

  const unsigned STRIDES_PER_WARP_ROUND = rows << 7;  // * 32 * 4
  unsigned c = threadIdx.x << 2;
  const size_t batch_row_index = (size_t)blockIdx.y * (rows * cols) + ((c & 0xffffffe0) * rows + (r << 5) + (c & 31));
  src += batch_row_index;
  dst += batch_row_index;
  for (unsigned index = 0; c < cols; c += 128, index += STRIDES_PER_WARP_ROUND) {
    char4 ch4 = *((const char4*)(src + index));
    sum += ((short)ch4.x + (short)ch4.y + (short)ch4.z + (short)ch4.w);
    square_sum = __dp4a(ch4, ch4, square_sum);
  }

  sum = WarpReduceSum<int32_t>(sum);
  square_sum = WarpReduceSum<int32_t>(square_sum);

  const float mean = (src_scale * sum / cols);
  const float rvar = rsqrtf(src_scale * src_scale * ((float)square_sum - ((float)sum * sum / cols)) / cols + epsilon);
  const __half2 mean2 = __float2half2_rn(mean);
  const __half2 var2 = __float2half2_rn(rvar);
  const __half2 src_scale2 = __float2half2_rn(src_scale);
  const __half2 dst_rscale2 = __float2half2_rn(1.0f / dst_scale);
  const __half4 zero4 = {__float2half2_rn(0.0f), __float2half2_rn(0.0f)};

  for (unsigned index = 0, c = threadIdx.x * 4; c < cols; c += 128, index += STRIDES_PER_WARP_ROUND) {
    char4 ch4 = __ldg((const char4*)(src + index));
    __half4 dqval4 = deqantize_char4_half4(ch4, src_scale2);
    const __half4 g4 = *((const __half4*)(gamma + c));
    const __half4 b4 = (beta == nullptr) ? zero4 : *((const __half4*)(beta + c));
    dqval4.xy = __hfma2(__hmul2(__hsub2(dqval4.xy, mean2), var2), g4.xy, b4.xy);
    dqval4.zw = __hfma2(__hmul2(__hsub2(dqval4.zw, mean2), var2), g4.zw, b4.zw);
    *(char4*)(dst + index) = quantize_half4_char4(dqval4, dst_rscale2);
  }
}

void QOrderLayerNorm(hipStream_t stream, const hipDeviceProp_t& /*device_prop*/,
                     const int8_t* src, const float src_scale, int8_t* dst, const float dst_scale,
                     const __half* gamma, const __half* beta, const float epsilon,
                     const unsigned batch, const unsigned rows, const unsigned cols) {
  dim3 threads(32, QORDER_LAYERNORM_ROWS_PER_BLOCK, 1);
  dim3 blocks((unsigned)(rows + QORDER_LAYERNORM_ROWS_PER_BLOCK - 1) / QORDER_LAYERNORM_ROWS_PER_BLOCK, (unsigned)batch, 1);
  QOrderLayerNormKernel<<<blocks, threads, 0, stream>>>(src, src_scale, dst, dst_scale, gamma, beta, epsilon, rows, cols);
}

// source matrix block 32 x 32, each thread handle 4 int8_t items,
// thread block size should be (8 cols_in_4, 32 rows, 1)
// grid size ((cols + 31) / 32, (rows + 31) / 32), batch)
__global__ void
ReorderS8RowToCol32Kernel(const int8_t* __restrict__ src, int8_t* __restrict__ dst, unsigned rows, unsigned cols) {
  unsigned int c = (blockIdx.x * blockDim.x + threadIdx.x) << 2;
  unsigned int r = blockIdx.y * blockDim.y + threadIdx.y;
  if (c < cols && r < rows) {
    const size_t batch_start = blockIdx.z * (rows * cols);
    const size_t src_index = batch_start + (r * cols + c);
    const size_t dst_index = batch_start + ((c & 0xffffffe0) * rows + (r << 5) + (c & 0x1f));
    *(char4*)(dst + dst_index) = *((const char4*)(src + src_index));
  }
}

void ReorderS8RowToCol32(hipStream_t stream, const hipDeviceProp_t& /* device_prop */,
                         const int8_t* src, int8_t* dst,
                         unsigned batch, unsigned rows, unsigned cols) {
  dim3 threads(8, 32, 1);
  dim3 blocks((unsigned)(cols / 32), (unsigned)((rows + 31) / 32), batch);
  ReorderS8RowToCol32Kernel<<<blocks, threads, 0, stream>>>(src, dst, rows, cols);
}

}  // namespace cuda
}  // namespace contrib
}  // namespace onnxruntime
